#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>          // CUDA FFT Libraries
#include <hip/hip_runtime_api.h>    // Helper functions for CUDA Error handling

 // OpenGL Graphics includes
#define HELPERGL_EXTERN_GL_FUNC_IMPLEMENTATION
#include <helper_gl.h>


// FluidsGL CUDA kernel definitions
#include "fluidsGL_kernels.cuh"


#define M_PI 3.14159265

// Particle data
extern GLuint vbo;                 // OpenGL vertex buffer object
extern struct hipGraphicsResource* cuda_vbo_resource; // handles OpenGL-CUDA exchange


__device__ inline cData empty()
{
	cData tmp = cData();
	tmp.x = tmp.y = tmp.z = 0;
	return tmp;
}


__device__ inline float getSquaredDistance(cData c1, cData c2=empty())
{
#ifdef Z_AXIS
	return (c1.x - c2.x) * (c1.x - c2.x) + (c1.y - c2.y) * (c1.y - c2.y) + (c1.z - c2.z) * (c1.z - c2.z);
#else
	return (c1.x - c2.x) * (c1.x - c2.x) + (c1.y - c2.y) * (c1.y - c2.y);
#endif // Z_AXIS

}

__device__ inline cData add(cData c1, cData c2)
{
	cData c3 = empty();
	c3.x = c1.x + c2.x;
	c3.y = c1.y + c2.y;
	c3.z = c1.z + c2.z;
	return c3;
}

__device__ inline cData add(cData c1, float m)
{
	cData c3 = empty();
	c3.x = c1.x + m;
	c3.y = c1.y + m;
#ifdef Z_AXIS
	c3.z = c1.z + m;
#else
	c3.z = c1.z;
#endif // Z_AXIS
	return c3;
}

__device__ inline cData subtract(cData c1, cData c2)
{
	cData c3 = empty();
	c3.x = c1.x - c2.x;
	c3.y = c1.y - c2.y;
	c3.z = c1.z - c2.z;
	return c3;
}

__device__ inline cData multiply(cData c1, cData c2)
{
	cData c3 = empty();
	c3.x = c1.x * c2.x;
	c3.y = c1.y * c2.y;
	c3.z = c1.z * c2.z;
	return c3;
}

__device__ inline cData multiply(cData c1, float m)
{
	cData c3 = empty();
	c3.x = c1.x * m;
	c3.y = c1.y * m;
	c3.z = c1.z * m;
	return c3;
}

__device__ inline cData divide(cData c1, float m)
{
	cData c3 = empty();
	c3.x = c1.x / m;
	c3.y = c1.y / m;
	c3.z = c1.z / m;
	return c3;
}

__device__ inline cData abs(cData c1)
{
	cData c3 = empty();
	c3.x = abs(c1.x);
	c3.y = abs(c1.y);
	c3.z = abs(c1.z);
	return c3;
}

__device__ cData setMagnitude(cData c, float m)
{
	float l = getSquaredDistance(c);
	if (l > 0.000001)
	{
		cData c1 = empty();
		c1 = multiply(c, m / sqrt(l));
		//printf("%f:%f:%f:::%f:%f:%f:::%f:%f:%f\n", l,getSquaredDistance(c1), m, c.x, c.y, c.z, c1.x, c1.y, c1.z);
		return c1;
	}
	return c;
}

__device__ cData limit(cData c, float m)
{
	float l = getSquaredDistance(c);
	if (l > m* m)
	{
		return setMagnitude(c, m);
	}
	return c;
}


__global__ void update_k(cData* part, cData* v, 
	int dx, float dt, int* grid_begin, int* grid_end,
	int* ids)
{
	int gtidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (gtidx < dx)
	{
		int fj = gtidx;

		cData pterm = part[2*fj];
		cData vterm = v[fj];
		cData fterm = empty();

		int x_grid = int(pterm.x * GRID_SIZE);
		int y_grid = int(pterm.y * GRID_SIZE);
		int z_grid = int(pterm.z * GRID_SIZE);


		//SHORE FUNCTIONS
		cData mid_cohesion = empty();
		cData mid_alignment = empty();
		cData mid_separation = empty();
		cData mid_obstacles = empty();
		int count_cohesion = 0;
		int count_alignment = 0;
		int count_separation = 0;
		
		for (int x = MAX(1, x_grid) - 1; x <= GRID_SIZE && x <= x_grid + 1; x++)
			for (int y = MAX(1, y_grid) - 1; y <= GRID_SIZE && y <= y_grid + 1; y++)
				for (int z = MAX(1, z_grid) - 1; z <= GRID_SIZE && z <= z_grid + 1; z++)
				{
					int grid_id = x + y * GRID_SIZE + z * GRID_SIZE * GRID_SIZE;
					for (int id = grid_begin[grid_id]; id < grid_end[grid_id]; id++)
					{
						int i = ids[id];
						if (i != fj && i>=0 && i<=dx)
						{
							float sqr_distance = getSquaredDistance(part[2 * i], pterm);

							//COHESION
							if (sqr_distance < SIGN_RADIUS * SIGN_RADIUS)
							{
								count_cohesion++;
								mid_cohesion = add(mid_cohesion, part[2 * i]);
							}

							//ALIGNMENT
							if (sqr_distance < SIGN_RADIUS * SIGN_RADIUS)
							{
								count_alignment++;
								mid_alignment = add(mid_alignment, v[i]);
							}

							//SEPARATION
							if (sqr_distance < SEPARATION_RADIUS * SEPARATION_RADIUS)
							{
								count_separation++;

								cData tmp = subtract(pterm, part[2 * i]);
								tmp = abs(tmp);
								tmp = multiply(tmp, -1);
								tmp = add(tmp, SEPARATION_RADIUS);

								if (sqr_distance < SEPARATION_RADIUS * SEPARATION_RADIUS / 100)
								{
									tmp = multiply(tmp, 3);
								}
								mid_separation.x += tmp.x * (pterm.x > part[2 * i].x ? 1 : -1);
								mid_separation.y += tmp.y * (pterm.y > part[2 * i].y ? 1 : -1);
								mid_separation.z += tmp.z * (pterm.z > part[2 * i].z ? 1 : -1);
							}
						}
					}
				}

		if (count_cohesion)
		{
			mid_cohesion = divide(mid_cohesion, count_cohesion);
			cData des = subtract(mid_cohesion, pterm);

			cData steer = limit(des, MAX_FORCE);
			steer = multiply(steer, COH_MULTI);

			fterm = add(fterm, steer);
		}
		if (count_alignment)
		{
			mid_alignment = divide(mid_alignment, count_alignment);

			cData steer = setMagnitude(mid_alignment, MAX_FORCE);
			steer = multiply(steer, ALI_MULTI);

			fterm = add(fterm, steer);
		}
		if (count_separation)
		{
			mid_separation = divide(mid_separation, count_separation);

			cData steer = setMagnitude(mid_separation, MAX_FORCE);
			steer = multiply(steer, SEP_MULTI);

			fterm = add(fterm, steer);
		}


		//AVOID OBSTACLES
		float r = OBSTACLES_RADIUS;
		if (pterm.x < 0.25 + r)
		{
			mid_obstacles.x += 0.25 + r - pterm.x;
		}
		if (pterm.y < 0.25 + r)
		{
			mid_obstacles.y += 0.25 + r - pterm.y;
		}
		if (pterm.z < 0.25 + r)
		{
#ifdef Z_AXIS
			mid_obstacles.z += 0.25 + r - pterm.z;
#endif // Z_AXIS
		}
		if (pterm.x > 0.75 - r)
		{
			mid_obstacles.x += (0.75 - pterm.x) - r;
		}
		if (pterm.y > 0.75 - r)
		{
			mid_obstacles.y += (0.75 - pterm.y) - r;
		}
		if (pterm.z > 0.75 - r)
		{
#ifdef Z_AXIS
			mid_obstacles.z += (0.75 - pterm.z) - r;
#endif // Z_AXIS
		}

		mid_obstacles = multiply(mid_obstacles, OBS_MULTI);
		fterm = add(fterm, mid_obstacles);


		//APPLY FORCES
		{
			cData tmp = multiply(fterm, dt);
			vterm = add(vterm, tmp);
			vterm = setMagnitude(vterm, MAX_SPEED);

			v[fj] = vterm;
		}


		//ADVERT PARTICLES
		{
			cData tmp = multiply(vterm, dt);
			tmp = add(tmp, 1.f);
			pterm = add(pterm, tmp);
			pterm.x = pterm.x - (int)pterm.x;
			pterm.y = pterm.y - (int)pterm.y;
			pterm.z = pterm.z - (int)pterm.z;

			float size_back = 0.02 * FISH_SIZE;

			cData v_scalled = setMagnitude(vterm, size_back);

			part[2 * fj] = pterm;
			part[2 * fj + 1] = subtract(pterm, v_scalled);
		}

	}
}

__global__ void get_grid_location_k(cData* part, int* ids, int* grid_ids, int dx)
{
	int gtidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (gtidx < dx)
	{
		int fj = gtidx;

		cData pterm = part[2 * fj];

		grid_ids[fj] = int(pterm.x * GRID_SIZE) + int(pterm.y * GRID_SIZE) * GRID_SIZE + int(pterm.z * GRID_SIZE) * GRID_SIZE * GRID_SIZE;
		ids[fj] = fj;
	}
}


__global__ void get_grid_boundries_k(int* grid_ids, int* grid_begin, int* grid_end, int dx)
{
	int gtidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (gtidx < dx)
	{
		int fj = gtidx;

		if (fj == 0)
			grid_begin[grid_ids[fj]] = fj;
		if (fj == dx - 1)
			grid_end[grid_ids[fj]] = fj+1;
		else
		{
			if (grid_ids[fj] != grid_ids[fj + 1])
			{
				grid_end[grid_ids[fj]] = fj+1;
				grid_begin[grid_ids[fj + 1]] = fj + 1;
			}
		}
	}
}

extern "C"
void advectParticles(GLuint vbo, cData * v, int* ids, int* grid_ids, int* grid_begin, int* grid_end, int dx, float dt)
{
	dim3 grid(dx/512, 1);
	dim3 tids(512, 1);

	cData* p;
	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	getLastCudaError("hipGraphicsMapResources failed");

	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&p, &num_bytes,
		cuda_vbo_resource);
	getLastCudaError("hipGraphicsResourceGetMappedPointer failed");


	get_grid_location_k << <grid, tids >>> (p, ids, grid_ids, dx);
	getLastCudaError("get_grid_location_k failed.");

	thrust::device_ptr<int> keys(grid_ids);
	thrust::device_ptr<int> values(ids);
	thrust::sort_by_key(keys, keys + dx, values);
	getLastCudaError("thrust sorting failed!");

	thrust::device_ptr<int> grid_begin_thrust(grid_begin);
	thrust::device_ptr<int> grid_end_thrust(grid_end);
	thrust::fill(grid_begin_thrust, grid_begin_thrust + GRID_SIZE* GRID_SIZE* GRID_SIZE, -1);
	thrust::fill(grid_end_thrust, grid_end_thrust + GRID_SIZE* GRID_SIZE* GRID_SIZE, -1);
	getLastCudaError("thrust sorting failed!");

	get_grid_boundries_k <<< grid, tids >>> (grid_ids, grid_begin, grid_end, GRID_SIZE * GRID_SIZE * GRID_SIZE);

	update_k << < grid, tids >> > (p, v, dx, dt, grid_begin, grid_end, ids);
	getLastCudaError("update_k failed.");


	hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	getLastCudaError("hipGraphicsUnmapResources failed");
}



extern "C"
void test(GLuint vbo, cData * v, int dx, float dt)
{
	float sum_milliseconds = 0;
	int times = 100;

	for (int i = 0; i < times; i++)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);

		//advectParticles(vbo, v, dx, dt);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		sum_milliseconds += milliseconds;
	}
	
	printf("Avarage time of update: %f", sum_milliseconds/times);
}